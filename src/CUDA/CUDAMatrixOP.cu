#include "CUDAMatrixOP.hpp"
#include "hip/hip_runtime_api.h"
#include <hipsparse.h>

#include <spdlog/spdlog.h>

// Now is only support `double`
void CUDAMatrixUTMU(PD::PDMatrix& U, PD::PDSparseMatrix& M, PD::PDMatrix& L)
{
    L.resize(U.cols(), U.cols());
    // Create CSC data
    int _nnz = M.nonZeros();
    std::vector<PD::PDScalar> entries(_nnz);
    std::vector<int> rowInds(_nnz);
    std::vector<int> colPtr(M.cols() + 1);
    int count = 0;
    colPtr[0] = 0;
    for (int k = 0; k < M.outerSize(); k++) {
        for (PD::PDSparseMatrix::InnerIterator it(M, k); it; ++it) {
            entries[count] = it.value();
            rowInds[count] = it.row();
            count++;
        }
        colPtr[k + 1] = count;
    }

    // Upload to GPU
    // - For SparseMatrix M
    PD::PDScalar* d_entries;
    int* d_rowInds;
    int* d_colPtr;
    checkCudaErrors(hipMalloc((void**)&d_entries, _nnz * sizeof(PD::PDScalar)));
    checkCudaErrors(hipMalloc((void**)&d_rowInds, _nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_colPtr, (M.cols() + 1) * sizeof(int)));
    checkCudaErrors(hipMemcpy(d_entries, entries.data(), _nnz * sizeof(PD::PDScalar), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rowInds, rowInds.data(), _nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_colPtr, colPtr.data(), (M.cols() + 1) * sizeof(int), hipMemcpyHostToDevice));
    // For DenseMatrix U
    PD::PDScalar* d_U;
    int U_N = U.rows();
    int U_M = U.cols();
    checkCudaErrors(hipMalloc((void**)&d_U, U.size() * sizeof(PD::PDScalar)));
    checkCudaErrors(hipMemcpy(d_U, U.data(), U.size() * sizeof(PD::PDScalar), hipMemcpyHostToDevice));
    // For DenseMatrix C
    PD::PDScalar* d_C;
    checkCudaErrors(hipMalloc((void**)&d_C, (M.rows() * U.cols()) * sizeof(PD::PDScalar)));
    // For Result DenseMatrix L
    PD::PDScalar* d_L;
    checkCudaErrors(hipMalloc((void**)&d_L, L.size() * sizeof(PD::PDScalar)));

    // Create Descripters
    hipsparseSpMatDescr_t matA = 0;
    hipsparseDnMatDescr_t matB = 0;
    hipsparseDnMatDescr_t matC = 0;
    hipsparseCreateCsc(&matA, M.rows(), M.cols(), _nnz, d_colPtr, d_rowInds,
        d_entries, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnMat(&matB, U.rows(), U.cols(), U.rows(), d_U, HIP_R_64F, HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&matC, M.rows(), U.cols(), M.rows(), d_C, HIP_R_64F, HIPSPARSE_ORDER_COL);

    //  - C = M * U <-> [M.rows(), U_M] = [M.rows(), M_M] * [M_M, U_M]
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);
    PD::PDScalar alpha = 1.0f;
    PD::PDScalar beta = 0.0f;
    size_t bufferSize;
    void* buffer = NULL;
    hipsparseSpMM_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA,
        matB, &beta, matC, HIP_R_64F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    auto state = hipsparseSpMM(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB,
        &beta, matC, HIP_R_64F, HIPSPARSE_SPMM_CSR_ALG1, buffer);
    // - L = U.transpose() * C <-> [U.cols(), U.cols()] = [U.cols(), N] * [N,
    // U.cols()]
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    hipblasDgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, U.cols(), U.cols(),
        M.rows(), &alpha, d_U, U.outerStride(), d_C, M.rows(), &beta,
        d_L, U.cols());

    checkCudaErrors(hipMemcpy(L.data(), d_L, L.size() * sizeof(PD::PDScalar), hipMemcpyDeviceToHost));

    // Free
    checkCudaErrors(hipFree(d_entries));
    checkCudaErrors(hipFree(d_rowInds));
    checkCudaErrors(hipFree(d_colPtr));
    checkCudaErrors(hipFree(d_U));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_L));
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
}

// Now is only support `double`
void MatrixUTSTMSU(PD::PDMatrix &U, PD::PDSparseMatrix &S, PD::PDSparseMatrix &M, PD::PDMatrix &L)
{
    spdlog::info(">>> TODO !!!");
    return ;
    // ST * M
    // Eigen sparse matrix deafult format is csc.
    // Malloc for M
    void* M_values = nullptr;
    void* M_rowInd = nullptr;
    void* M_colPtr = nullptr;
    checkCudaErrors(hipMalloc(&M_values, sizeof(PD::PDScalar) * M.nonZeros()));
    checkCudaErrors(hipMalloc(&M_rowInd, sizeof(int) * M.nonZeros()));
    checkCudaErrors(hipMalloc(&M_colPtr, sizeof(int) * (M.cols() + 1)));
    checkCudaErrors(hipMemcpy(M_values, M.valuePtr(), sizeof(PD::PDScalar) * M.nonZeros(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(M_rowInd, M.innerIndexPtr(), sizeof(int) * M.nonZeros(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(M_colPtr, M.outerIndexPtr(), sizeof(int) * (M.cols() + 1), hipMemcpyHostToDevice));
    hipsparseSpMatDescr_t M_mat = 0;
    hipsparseDnVecDescr_t M_vecX = 0;
    hipsparseDnVecDescr_t M_vecY = 0;
    checkCudaErrors(hipsparseCreateCsc(&M_mat, M.rows(), M.cols(), M.nonZeros(), M_colPtr,
        M_rowInd, M_values, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // Malloc for S
    void* S_values = nullptr;
    void* S_rowInd = nullptr;
    void* S_colPtr = nullptr;
    checkCudaErrors(hipMalloc(&S_values, sizeof(PD::PDScalar) * S.nonZeros()));
    checkCudaErrors(hipMalloc(&S_rowInd, sizeof(int) * S.nonZeros()));
    checkCudaErrors(hipMalloc(&S_colPtr, sizeof(int) * (S.cols() + 1)));
    checkCudaErrors(hipMemcpy(S_values, S.valuePtr(), sizeof(PD::PDScalar) * S.nonZeros(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(S_rowInd, S.innerIndexPtr(), sizeof(int) * S.nonZeros(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(S_colPtr, S.outerIndexPtr(), sizeof(int) * (S.cols() + 1), hipMemcpyHostToDevice));
    hipsparseSpMatDescr_t S_mat = 0;
    hipsparseDnVecDescr_t S_vecX = 0;
    hipsparseDnVecDescr_t S_vecY = 0;
    checkCudaErrors(hipsparseCreateCsc(&S_mat, M.rows(), M.cols(), M.nonZeros(), S_colPtr,
        S_rowInd, S_values, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    
    // todo...
    // hipsparseSpGEMM_compute(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void *alpha, hipsparseConstSpMatDescr_t matA, hipsparseConstSpMatDescr_t matB, const void *beta, hipsparseSpMatDescr_t matC, hipDataType computeType, hipsparseSpGEMMAlg_t alg, hipsparseSpGEMMDescr_t spgemmDescr, size_t *bufferSize2, void *externalBuffer2)
}