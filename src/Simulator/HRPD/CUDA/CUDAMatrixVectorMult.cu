#include "hip/hip_runtime.h"
#include "CUDAMatrixVectorMult.hpp"
#include "hip/hip_runtime_api.h"
#include <spdlog/spdlog.h>


__global__ void doubleToFloatMemCpyKernel(int n, int coord, double* source, float* target)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        target[i * 3 + coord] = source[i];
    }
}

__global__ void doubleToFloatMemCpyKernel(int n, int coord, float* source, float* target)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        target[i * 3 + coord] = source[i];
    }
}

__global__ void elementWiseMultiplyKernel(int n, double* a, double* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        a[i] = a[i] * b[i];
    }
}

__global__ void elementWiseMultiplyKernel(int n, float* a, float* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        a[i] = a[i] * b[i];
    }
}

void doubleToFloatDeviceCpy(int n, int coord, double* source, float* target)
{
    doubleToFloatMemCpyKernel<<<(n + 255) / 256, 256>>>(n, coord, source, target);
}

void doubleToFloatDeviceCpy(int n, int coord, float* source, float* target)
{
    doubleToFloatMemCpyKernel<<<(n + 255) / 256, 256>>>(n, coord, source, target);
}

void elementWiseMultiply(int n, double* a, double* b)
{
    elementWiseMultiplyKernel<<<(n + 255) / 256, 256>>>(n, a, b);
}

void elementWiseMultiply(int n, float* a, float* b)
{
    elementWiseMultiplyKernel<<<(n + 255) / 256, 256>>>(n, a, b);
}

///////////////////////////////////////////////////////////////////////////////////////////
//  Struct CUDAMatrixVectorMultiplier
///////////////////////////////////////////////////////////////////////////////////////////
CUDAMatrixVectorMultiplier::CUDAMatrixVectorMultiplier(PD::PDMatrix& mat)
{
    if (cublasLibInitialized == false) {
        hipblasCreate(&cublasLibHandle);
        cublasLibInitialized = true;
    }

    m_numRows = mat.rows();
    m_numCols = mat.cols();

    checkCudaErrors(hipMalloc((void**)&m_cudaInVec, sizeof(PD::PDScalar) * m_numCols));
    checkCudaErrors(hipMalloc((void**)&m_cudaMat, sizeof(PD::PDScalar) * m_numRows * m_numCols));
    checkCudaErrors(hipMalloc((void**)&m_cudaOutVec, sizeof(PD::PDScalar) * m_numRows));
    // If the matrix has more columns then rows, we need to upload it transposed,
    // since hipblasDgemv expects a matrix with more rows than columns...
    if (m_numRows < m_numCols) {
        PD::PDMatrix matT = mat.transpose();
        const void* matDataPointer = (const void*)matT.data();
        checkCudaErrors(hipblasSetMatrix(m_numCols, m_numRows, sizeof(PD::PDScalar), matDataPointer, m_numCols, (void*)m_cudaMat, m_numCols));
    }
    else {
        const void* matDataPointer = (const void*)mat.data();
        checkCudaErrors(hipblasSetMatrix(m_numRows, m_numCols, sizeof(PD::PDScalar), matDataPointer, m_numRows, (void*)m_cudaMat, m_numRows));
    }
}

CUDAMatrixVectorMultiplier::CUDAMatrixVectorMultiplier(PD::PDMatrix& mat, PD::PDVector& masses)
    : CUDAMatrixVectorMultiplier(mat)
{
    m_massesSize = masses.rows();
    checkCudaErrors(hipMalloc((void**)&m_cudaMassesVec, sizeof(PD::PDScalar) * m_massesSize));
    checkCudaErrors(hipblasSetVector(m_massesSize, sizeof(PD::PDScalar), masses.data(), 1, (void*)(m_cudaMassesVec), 1));
    hipDeviceSynchronize();
}

CUDAMatrixVectorMultiplier::~CUDAMatrixVectorMultiplier()
{
    hipFree(m_cudaMat);
    hipFree(m_cudaInVec);
    hipFree(m_cudaOutVec);
    if (!m_cudaMassesVec) hipFree(m_cudaMassesVec);
}

void CUDAMatrixVectorMultiplier::mult(const void* inData, void* outData, PD::PDScalar& alpha, bool transpose, int coord, int cutoff)
{
    if (!transpose) {
        checkCudaErrors(hipblasSetVector(m_numCols, sizeof(PD::PDScalar), inData, 1, (void*)(m_cudaInVec), 1));
        if (m_massesSize == m_numCols && m_cudaMassesVec) {
            elementWiseMultiply(m_numCols, m_cudaInVec, m_cudaMassesVec);
        }
    }
    else {
        checkCudaErrors(hipblasSetVector(m_numRows, sizeof(PD::PDScalar), inData, 1, (void*)(m_cudaOutVec), 1));
        if (m_massesSize == m_numRows && m_cudaMassesVec) {
            elementWiseMultiply(m_numRows, m_cudaOutVec, m_cudaMassesVec);
        }
    }
    if (m_numRows < m_numCols) {
        if (!transpose) {
            // In this case a product with the untransposed matrix is desired, however, the matrix stored
            // on the GPU has been transposed before (since it had more columns than rows), so the operation
            // for Dgemv should be OP_T.
            // On the other hand the pointers to cudaInVec and cudaOutVec have the correct sizes (M and N
            // respectively) so that they appear in the normal order.
            // The reasoning for the other three cases below can be deduced from this example.
            checkCudaErrors(hipblasDgemv(cublasLibHandle, HIPBLAS_OP_T, m_numCols, m_numRows, &alpha, m_cudaMat, m_numCols, m_cudaInVec, 1, &cublasZero, m_cudaOutVec, 1));
        }
        else {
            checkCudaErrors(hipblasDgemv(cublasLibHandle, HIPBLAS_OP_N, m_numCols, m_numRows, &alpha, m_cudaMat, m_numCols, m_cudaOutVec, 1, &cublasZero, m_cudaInVec, 1));
        }
    }
    else {
        if (!transpose) {
            checkCudaErrors(hipblasDgemv(cublasLibHandle, HIPBLAS_OP_N, m_numRows, m_numCols, &alpha, m_cudaMat, m_numRows, m_cudaInVec, 1, &cublasZero, m_cudaOutVec, 1));
        }
        else {
            checkCudaErrors(hipblasDgemv(cublasLibHandle, HIPBLAS_OP_T, m_numRows, m_numCols, &alpha, m_cudaMat, m_numRows, m_cudaOutVec, 1, &cublasZero, m_cudaInVec, 1));
        }
    }

    if (outData) {
        if (!transpose) {
            checkCudaErrors(hipMemcpy(outData, m_cudaOutVec, sizeof(PD::PDScalar) * m_numRows, hipMemcpyDeviceToHost));
        }
        else {
            checkCudaErrors(hipMemcpy(outData, m_cudaInVec, sizeof(PD::PDScalar) * m_numCols, hipMemcpyDeviceToHost));
        }
    }

    // BUFFER MAP
    if (bufferInitifalized) {
        // if (!m_glArrayPtr) {
        //     // Hi-jack the buffer from OpenGL
        //     hipGraphicsResource_t res;
        //     glBindBuffer(GL_ARRAY_BUFFER, m_glbufferId);
        //     checkCudaErrors(hipGraphicsGLRegisterBuffer(&res, m_glbufferId, hipGraphicsRegisterFlagsNone));
        //     checkCudaErrors(hipGraphicsMapResources(1, &res, 0));
        //     size_t size;
        //     checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&m_glArrayPtr, &size, res));
        //     glBindBuffer(GL_ARRAY_BUFFER, 0);
        // }
        // if (m_glArrayPtr) {
        //     //Copy from m_cudaOutVec to m_glArrayPtr while casting from PDScalar to float
        //     int N = m_numRows;
        //     if (cutoff >= 0) {
        //         N = cutoff;
        //     }
        //     doubleToFloatDeviceCpy(N, coord, m_cudaOutVec, m_glArrayPtr);
        // }

        //Copy from m_cudaOutVec to m_glArrayPtr while casting from PDScalar to float
        int N = m_numRows;
        if (cutoff >= 0) {
            N = cutoff;
        }
        checkCudaErrors(hipGraphicsMapResources(1, &m_res, 0));
        size_t size;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&m_glArrayPtr, &size, m_res));
        doubleToFloatDeviceCpy(N, coord, m_cudaOutVec, m_glArrayPtr);
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipGraphicsUnmapResources(1, &m_res, 0));
    }

    hipDeviceSynchronize();
}

void CUDAMatrixVectorMultiplier::initBufferMap(GLuint bufferId)
{
    spdlog::info(">>> CUDAMatrixVectorMultiplier::initBufferMap()");
    m_glbufferId = bufferId;
    glBindBuffer(GL_ARRAY_BUFFER, m_glbufferId);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&m_res, m_glbufferId, hipGraphicsRegisterFlagsNone));
    bufferInitifalized = true;
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    spdlog::info(">>> CUDAMatrixVectorMultiplier::initBufferMap() - After");
}

///////////////////////////////////////////////////////////////////////////////////////////
//  Struct CUDASparseMatrixVectorMultiplier
///////////////////////////////////////////////////////////////////////////////////////////
CUDASparseMatrixVectorMultiplier::CUDASparseMatrixVectorMultiplier(PD::PDSparseMatrix& mat)
{
    if (!cusparseLibInitialized) {
        hipsparseCreate(&cusparseLibHandle);
        cusparseLibInitialized = true;
    }
    m_numRows = mat.rows(), m_numCols = mat.cols();
    m_nnz = mat.nonZeros();
    checkCudaErrors(hipMalloc((void**)&m_cudaInVec, sizeof(PD::PDScalar) * m_numCols));
    checkCudaErrors(hipMalloc((void**)&m_cudaOutVec, sizeof(PD::PDScalar) * m_numRows));

    // Create the CSC data for the sparse matrix
    PD::PDScalar* entries = new PD::PDScalar[m_nnz];
    int* rowInds = new int[m_nnz];
    int* colPtr = new int[m_numCols + 1];
    unsigned int counter = 0;
    colPtr[0] = 0;
    for (int k = 0; k < mat.outerSize(); ++k) {
        for (PD::PDSparseMatrix::InnerIterator it(mat, k); it; ++it) {
            entries[counter] = it.value();
            rowInds[counter] = it.row();
            counter++;
        }
        colPtr[k + 1] = counter;
    }

    // Upload the CSC data
    checkCudaErrors(hipMalloc((void**)&m_cudaMatData, sizeof(PD::PDScalar) * m_nnz));
    checkCudaErrors(hipMemcpy(m_cudaMatData, entries, sizeof(PD::PDScalar) * m_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&m_cudaRowInd, sizeof(int) * m_nnz));
    checkCudaErrors(hipMemcpy(m_cudaRowInd, rowInds, sizeof(int) * m_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&m_cudaColPtr, sizeof(int) * (m_numCols + 1)));
    checkCudaErrors(hipMemcpy(m_cudaColPtr, colPtr, sizeof(int) * (m_numCols + 1), hipMemcpyHostToDevice));

    // Set up description
    checkCudaErrors(hipsparseCreateMatDescr(&m_desc));
    checkCudaErrors(hipsparseSetMatType(m_desc, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(m_desc, HIPSPARSE_INDEX_BASE_ZERO));
}

void CUDASparseMatrixVectorMultiplier::mult(const void* inData, void* outData, PD::PDScalar& alpha)
{
    checkCudaErrors(hipblasSetVector(m_numCols, sizeof(PD::PDScalar), inData, 1, (void*)(m_cudaInVec), 1));
    checkCudaErrors(hipMemcpy(m_cudaInVec, inData, sizeof(PD::PDScalar) * m_numCols, hipMemcpyHostToDevice));

    /*
        NOTE:
        We do NOT multiply with the transpose of the matrix that was passed to the constructor!!!
        The reason that we set the operation to transpose is that cuSparse only offers a multiplication
        for compressed sparse row (CSR) format type matrices, but the matrix mat is in compressed sparse
        column (CSC) format.
        However, using the CSC description of mat as a CSR description, leads to the transpose of mat.
        Thus, multiplying with the transpose in CSR amounts to multiplying with the non-transposed in
        CSC.
    */
    hipsparseSpMatDescr_t matA = 0;
    checkCudaErrors(
        hipsparseCreateCsc(&matA, m_numCols, m_numRows, m_nnz,
            m_cudaColPtr, m_cudaRowInd, m_cudaMatData,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    hipsparseDnVecDescr_t vecX = 0;
    hipsparseDnVecDescr_t vecY = 0;
    checkCudaErrors(hipsparseCreateDnVec(&vecX, m_numRows, m_cudaInVec, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecY, m_numCols, m_cudaOutVec, HIP_R_64F));

    checkCudaErrors(
        hipsparseSpMV(cusparseLibHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &m_alpha, matA, vecX, &m_zero, vecY, HIP_R_64F,
            HIPSPARSE_SPMV_ALG_DEFAULT, NULL));
    checkCudaErrors(hipMemcpy(outData, m_cudaOutVec, sizeof(PD::PDScalar) * m_numRows, hipMemcpyDeviceToHost));
}